
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void parallel_vector_add(int* d_a, int* d_b, int* d_c, int* d_n)
{
	int i = (blockIdx.x*blockDim.x)+threadIdx.x;
	//printf("I am thread #%d.", i);
	if(i < *d_n){
		printf("I am thread #%d. and about to compute c[%d].\n", i, i);
		d_c[i] = d_a[i]+d_b[i];
	}
	else {
		printf("I am thread #%d. and doing nothing.\n", i);
	}
}

int main() {

	int n;

	scanf("%d", &n);
	
	// declare input and output on host
	int h_a[n];
	int h_b[n];

	for(int i=0; i<n; i++)
	{
		h_a[i] = i;
		h_b[i] = n-i;
	}


	int h_c[n];

	// PART I: Copy data from host to device
	int* d_a, *d_b, *d_c, *d_n;
	hipMalloc((void **) &d_a, n*sizeof(int));
	hipMalloc((void **) &d_b, n*sizeof(int));
	hipMalloc((void **) &d_c, n*sizeof(int));
	hipMalloc((void **) &d_n, sizeof(int));

	hipMemcpy(d_a, &h_a, n*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_b, &h_b, n*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	// PART II: Kernel launch
	int nb = (n%10>1)? (n/10)+1:(n/10);
	parallel_vector_add<<<nb, 10>>>(d_a, d_b, d_c, d_n);
	hipDeviceSynchronize();

	// PART III: Copy data from device back to host. and free all data allocate on device
	hipMemcpy(&h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for(int i=0; i<n; i++)
		printf("%d ", h_c[i]);

	printf("\n");

}
