
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 32

void add(int *X, int *Y, int *Z){
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			Z[i*N+j] = X[i*N+j] + Y[i*N+j] ;
		}
	}
}

__global__ void add_kernel(int *X, int *Y, int *Z){
	int i = threadIdx.x ;
	int j = threadIdx.y ;
	
	if(i < N && j < N){
		Z[i*N+j] = X[i*N+j] + Y[i*N+j] ;
	}
}

int main(){
	
	int X[N*N] ;
	int Y[N*N] ;

	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			X[i*N+j] = 0 ;
			Y[i*N+j] = 1 ;
		}
	}

	int Z[N*N] ;

	int *d_X, *d_Y, *d_Z ;
	hipMalloc((void **) &d_X, (N*N)*sizeof(int)) ;
	hipMalloc((void **) &d_Y, (N*N)*sizeof(int)) ;
	hipMalloc((void **) &d_Z, (N*N)*sizeof(int)) ;

	hipMemcpy(d_X, &X, (N*N)*sizeof(int), hipMemcpyHostToDevice) ;
	hipMemcpy(d_Y, &Y, (N*N)*sizeof(int), hipMemcpyHostToDevice) ;

	dim3 dimGrid(32,32,1) ;
	dim3 dimBlock(32,32,1) ;

	hipEvent_t start, stop ;
	hipEventCreate(&start) ;
	hipEventCreate(&stop) ;

	hipEventRecord(start) ;
	add_kernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_Z) ;
	//add(X, Y, Z) ;
	hipEventRecord(stop) ;

	hipMemcpy(&Z, d_Z, (N*N)*sizeof(int), hipMemcpyDeviceToHost) ;

	hipEventSynchronize(stop) ;
	float millisec = 0 ;
	hipEventElapsedTime(&millisec, start, stop) ;

	hipFree(d_X) ;
	hipFree(d_Y) ;
	hipFree(d_Z) ;

	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			printf("%d ", Z[i*N+j]) ;
		}
		printf("\n") ;
	}
	
	printf("Time : %lf\n", millisec) ;
}