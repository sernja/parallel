
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#define N 1000000
#define R 3
#define BLOCK_SIZE 512

__global__ void singlethread_stencil(int* d_in, int* d_out, int M)
{
	int tid = threadIdx.x;
	if(tid == 0){
		for(int i=0; i<M; i++){
			for(int j=-R; j<=R; j++)
				d_out[i] += d_in[i+R+j];
		}
	}

}

__global__ void multiplethreads_stencil(int* d_in, int* d_out, int M)
{
	int tid = blockIdx.x*BLOCK_SIZE+ threadIdx.x;
	if(tid <M){
		int result = 0;
		for(int j=-R; j<=R; j++)
			result += d_in[tid+R+j];
		d_out[tid] = result;
	}
}

__global__ void faster_stencil(int* d_in, int* d_out, int M)
{
	__shared__ int temp[BLOCK_SIZE + (2*R)];

	int g_id = blockIdx.x*BLOCK_SIZE+ threadIdx.x;
	int l_id = threadIdx.x + R;

	if(g_id < M){

		temp[l_id] = d_in[g_id+R];
		if(threadIdx.x < R){
			temp[l_id-R] = d_in[g_id];
			temp[l_id+BLOCK_SIZE] = d_in[g_id+BLOCK_SIZE];
		}
	}

	__syncthreads();

	int result = 0;

	for(int j=-R; j<=R; j++)
		result+= temp[l_id+R];
	d_out[g_id] = result;

}

int main()
{
	int M = N-2*R;

	int h_in[N];
	int h_out[M];

	for(int i=0; i < N; i++)
		h_in[i] = 1;

	int* d_in;
	int* d_out;


	//Part 1: Memory transfer from host to device
	hipMalloc((void**) &d_in, N*sizeof(int));
	hipMalloc((void**) &d_out, M*sizeof(int));

	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);

	//Part 2: Execute kernel

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	//singlethread_stencil<<<1, BLOCK_SIZE>>>(d_in, d_out, M);
	//faster_stencil<<<(int) ceil(M/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);
	multiplethreads_stencil<<<(int) ceil(M/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	//Part 3: memory tranfer from device to host
	hipMemcpy(&h_out, d_out, M*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	//Part 4: Check the result

	for(int i=0; i<M; i++){
		if(h_out[i] != 2*R+1){
			printf("Incorrent result.\n");
			return -1;
		}
	}
	printf("Correct result!\n");
	printf("time = %f\n", milliseconds);


}
